
#include <hip/hip_runtime.h>
//TEST(smoke):COMPILE: -pass-through nvrtc -target ptx -entry hello tests/cuda/cuda-compile.cu

__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}
